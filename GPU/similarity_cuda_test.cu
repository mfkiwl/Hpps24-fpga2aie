#include "hip/hip_runtime.h"
#include "common.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <chrono>

#define GMEM 0 // 1 to force the use of GMEM over SMEM

// very slow, but needed to simulate the data passing from host to device
void generate_matrix(uint8_t *matrix, int rows, int columns) {
    for (int i = 0; i < columns*rows; i++) {
        matrix[i] = i  % SIZE;
    }
}

// compute joint histogram using global memory
__global__ void histogram_gmem(int height, int width, uint8_t *img1, uint8_t *img2, uint32_t *out) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Define offset for each thread block
    long offset = blockIdx.y * gridDim.x * SIZE * SIZE + blockIdx.x * SIZE * SIZE;

    int numThreads = gridDim.x * blockDim.x * gridDim.y * blockDim.y;

    int threadId = y * width + x;
    int img_size = height * width;
    int iterations = (img_size + numThreads - 1) / numThreads;

    int idx = threadId;

    for (int i = 0; i < iterations; i++) {
        if (idx >= width * height) return;

        uint8_t a = img1[idx];
        uint8_t b = img2[idx];

        // Update the histogram using atomics
        atomicAdd(&out[offset + a * SIZE + b], 1);

        idx += numThreads; //update index
    }
}

#if SIZE <= 128
__global__ void histogram_smem(int height, int width, uint8_t *img1, uint8_t *img2, uint32_t *out) {
    // Define constants and indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    const int SIZE = 256;  // Assuming the size of each dimension of the histogram
    int threadId = y * width + x;
    int img_size = height * width;
    int numThreads = gridDim.x * blockDim.x * gridDim.y * blockDim.y;
    int iterations = (img_size + numThreads - 1) / numThreads;

    // Define offset for the current block's histogram in global memory
    long offset = blockIdx.y * gridDim.x * SIZE * SIZE + blockIdx.x * SIZE * SIZE;

    // Allocate shared memory for the histogram
    __shared__ uint32_t hist_shared[SIZE][SIZE];
    
    // Initialize shared memory histogram to zero
    for (int i = threadIdx.y; i < SIZE; i += blockDim.y) {
        for (int j = threadIdx.x; j < SIZE; j += blockDim.x) {
            hist_shared[i][j] = 0;
        }
    }
    __syncthreads();

    // Update the shared memory histogram
    for (int i = 0; i < iterations; i++) {
        int idx = threadId + i * numThreads;
        if (idx >= width * height) continue;

        uint8_t a = img1[idx];
        uint8_t b = img2[idx];

        // Increment the shared histogram using atomic operations
        atomicAdd(&hist_shared[a][b], 1);
    }
    __syncthreads();

    // Write the shared histogram back to global memory
    for (int i = threadIdx.y; i < SIZE; i += blockDim.y) {
        for (int j = threadIdx.x; j < SIZE; j += blockDim.x) {
            atomicAdd(&out[offset + i * SIZE + j], hist_shared[i][j]);
        }
    }
}
#endif

// reduce n instances of joint histograms to a single joint histogram
__global__ void reduce_add(uint32_t *histograms, uint32_t *reduced_histo, int instances) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= SIZE || y >= SIZE) return;

    int result_idx = x * SIZE + y;
    uint32_t sum = 0;

    for (int i = 0; i < instances; i++) {
        int offset = i * SIZE * SIZE;
        sum += histograms[offset + result_idx];
        __syncthreads();
    }

    reduced_histo[result_idx] = sum;
}

// Reduce by row. Call on SIZE threads
__global__ void reduce_by_row(uint32_t *histo, uint32_t *marginal) {
    int row = threadIdx.x;

    if (row >= SIZE) return;
    uint32_t sum = 0;
    // Reduce by summing over each column in the row
    for (int col = 0; col < SIZE; ++col) {
        sum += histo[row * SIZE + col];
    }

    // Store the result for this row
    marginal[row] = sum;
}

// Reduce by column. Call on SIZE threads
__global__ void reduce_by_column(uint32_t *histo, uint32_t *marginal) {
    int col = threadIdx.x;

    if (col >= SIZE) return;

    uint32_t sum = 0;
    // Reduce by summing over each row in the column
    for (int row = 0; row < SIZE; ++row) {
        sum += histo[row * SIZE + col];
    }

    // Store the result for this column
    marginal[col] = sum;
}

__device__ float entropy(uint32_t *histogram_row, int img_size) {
    float e = 0.0;
    for (int i = 0; i < SIZE; i++) {
        float p = ((float)histogram_row[i]) / img_size;
        if (p > 0.0f)
            e -= p * log2(p);
    }
    return e;
}

__global__ void entropies(uint32_t *histogram, float *histo_entropy, int img_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= SIZE+2) return;

    uint32_t *my_row = &histogram[SIZE * idx];
    histo_entropy[idx] = entropy(my_row, img_size);
}

#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

// Error checking function
inline void checkCudaError(const char* msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

int main(int argc, char *argv[]){

    // Generate matrices

    uint8_t* img1_host = (uint8_t*) malloc(MIN_IMG_SIZE * MAX_DEPTH * sizeof(uint8_t));
    uint8_t* img2_host = (uint8_t*) malloc(MIN_IMG_SIZE * MAX_DEPTH * sizeof(uint8_t));

    generate_matrix(img1_host, MIN_IMG_SIZE, MAX_DEPTH);
    generate_matrix(img2_host, MIN_IMG_SIZE, MAX_DEPTH);

    int max_histo_grid = 8;

    FILE *csv_file = fopen("results_cuda.csv", "w");
    if (csv_file == NULL) {
        perror("Failed to open CSV file");
        return 1;
    }

    // Write CSV header
    fprintf(csv_file, "ImageSize,Transfer,Histogram,Gather,Reduce,Entropy\n");

    for (int depth = MAX_DEPTH; depth <= MAX_DEPTH; depth++) {
        for(int i = 0; i < 100; i++){

            int rows = MIN_IMG_SIZE;
            int columns = depth;

            // Move matrices to device
            auto start_time = std::chrono::high_resolution_clock::now();

            // Allocate space for matrices on device
            uint8_t *matrix1_device, *matrix2_device;
            hipMalloc((void**)&matrix1_device, rows * columns * sizeof(uint8_t));
            //checkCudaError("hipMalloc for matrix1_device");

            hipMalloc((void**)&matrix2_device, rows * columns * sizeof(uint8_t));
            //checkCudaError("hipMalloc for matrix2_device");

            hipMemcpy(matrix1_device, img1_host, rows * columns * sizeof(uint8_t), hipMemcpyHostToDevice);
            //checkCudaError("hipMemcpy matrix1_device");

            hipMemcpy(matrix2_device, img2_host, rows * columns * sizeof(uint8_t), hipMemcpyHostToDevice);
            //checkCudaError("hipMemcpy matrix2_device");

            auto end_time = std::chrono::high_resolution_clock::now();
            auto dt_to_device = end_time - start_time;
            //printf("Time to move to device: %lf ms\n", dt_to_device.count());

            // Compute histograms

            start_time = std::chrono::high_resolution_clock::now();

            dim3 blockDim(32, 32);

            int gridx = ((   rows + blockDim.x - 1) / blockDim.x < max_histo_grid) ? (   rows + blockDim.x - 1) / blockDim.x : max_histo_grid;
            int gridy = ((columns + blockDim.y - 1) / blockDim.y < max_histo_grid) ? (columns + blockDim.y - 1) / blockDim.y : max_histo_grid;

            dim3 gridDim(gridx, gridy);

            int blocks = gridDim.x * gridDim.y;

            uint32_t *multi_histo, *histo;

            long histos_size = SIZE * SIZE;
            histos_size = histos_size * blocks * sizeof(uint32_t);

            // Allocate space for histograms on device

            hipMalloc((void**)&multi_histo, histos_size);
            //checkCudaError("hipMalloc for multi_histo");

            hipMalloc((void**)&histo, HISTO_SIZE * sizeof(uint32_t));
            //checkCudaError("hipMalloc for histo");

            hipMemset(multi_histo, 0, SIZE * SIZE * blocks* sizeof(uint32_t));
            //checkCudaError("hipMemset for multi_histo");

            hipMemset(histo, 0, HISTO_SIZE * sizeof(uint32_t));
            //checkCudaError("hipMemset for histo");

            if(verbose){
                printf("------------------------\n");
                printf("Image SIZE: %d x %d\n", rows, columns );
                printf("Joint histogram SIZE: %d x %d\n", SIZE, SIZE);
                printf("------------------------\n");
                printf("Block Size: %d, %d\n", blockDim.x, blockDim.y);
                printf("Grid:       %d, %d\n", gridDim.x, gridDim.y);
                printf("Thread per block: %d\n", blockDim.x * blockDim.y);
                printf("Blocks per grid: %d\n", gridDim.x * gridDim.y);
                printf("------------------------\n");
            }

            hipDeviceSynchronize();
            
            // Call histogram kernel
            #if GMEM || SIZE > 128 
                histogram_gmem<<<gridDim, blockDim>>>(rows, columns, matrix1_device, matrix2_device, multi_histo);
                //checkCudaError("histogram_gmem_atomics kernel launch");
            #else
                histogram_smem<<<gridDim, blockDim, SIZE * SIZE * sizeof(uint32_t)>>>(rows, columns, matrix1_device, matrix2_device, multi_histo);
                //checkCudaError("histogram_smem_atomics kernel launch");
            #endif

            hipDeviceSynchronize();

            end_time = std::chrono::high_resolution_clock::now();
            auto histo_time = end_time - start_time;

            //printf("Time to compute histograms: %lf ms\n", histo_time.count());

            start_time = std::chrono::high_resolution_clock::now();

            dim3 threadsPerBlock(32, 32);
            dim3 blocksPerGrid(SIZE / 32, SIZE / 32);

            reduce_add<<<blocksPerGrid, threadsPerBlock>>>(multi_histo, histo, blocks);
            //checkCudaError("reduce_add kernel launch");

            if(verbose){
                printf("------------------------\n");
                printf("Block: %d, %d\n", threadsPerBlock.x, threadsPerBlock.y);
                printf("Grid:  %d, %d\n", blocksPerGrid.x, blocksPerGrid.y);
                printf("Blocks per grid: %d\n", blocksPerGrid.x * blocksPerGrid.y);
                printf("------------------------\n");
            }

            //marginals

            uint32_t* marginal_x = &histo[SIZE*SIZE];
            uint32_t* marginal_y = &histo[SIZE*(SIZE+1)];

            hipDeviceSynchronize();

            end_time = std::chrono::high_resolution_clock::now();
            auto gather_time = end_time - start_time;
            
            //printf("Time to gather: %lf ms\n", gather_time.count());

            start_time = std::chrono::high_resolution_clock::now();

            reduce_by_row<<<1, SIZE>>>(histo, marginal_x);
            //checkCudaError("reduce_by_row launch");

            reduce_by_column<<<1, SIZE>>>(histo, marginal_y);
            //checkCudaError("reduce_by_column launch");

            end_time = std::chrono::high_resolution_clock::now();
            auto reduce_time = end_time - start_time;
            //printf("Time compute reduce: %lf ms\n", reduce_time.count());

            // entropy

            start_time = std::chrono::high_resolution_clock::now();

            float *histo_entropy_device;
            hipMalloc((void**)&histo_entropy_device, (SIZE + 2) * sizeof(float));
            //checkCudaError("hipMalloc for histo_entropy_device");

            hipDeviceSynchronize();
            entropies<<<1, SIZE+2>>>(histo, histo_entropy_device, rows * columns);
            //checkCudaError("entropies kernel launch");

            float *histo_entropy_host = (float *)malloc( (SIZE+2) * sizeof(float));
            hipDeviceSynchronize();
            hipMemcpy(histo_entropy_host, histo_entropy_device, (SIZE+2) * sizeof(float), hipMemcpyDeviceToHost);
            //checkCudaError("hipMemcpy histo_entropy_host");

            float Hxy = 0.0;
            for (int i = 0; i < SIZE; i++) {
                Hxy += histo_entropy_host[i];
            }
            float Hx = histo_entropy_host[SIZE];
            float Hy = histo_entropy_host[SIZE+1];

            end_time = std::chrono::high_resolution_clock::now();
            auto entropy_time = end_time - start_time;

            //printf("Time to compute H(X;Y):  %lf ms\n", entropy_time.count());

            printf("------------------------\n");
            printf("H(X;Y) : %.8f\n", Hxy);
            printf("H(X)   : %.8f\n", Hx);
            printf("H(Y)   : %.8f\n", Hy);
            printf("MI(X;Y): %.8f\n", Hx+Hy-Hxy);
            printf("------------------------\n");

            hipFree(matrix1_device);
            hipFree(matrix2_device);
            hipFree(histo);
            hipFree(multi_histo);
            hipFree(histo_entropy_device);

            double transfer_ms   = std::chrono::duration<double, std::milli>(dt_to_device).count();
            double histo_ms      = std::chrono::duration<double, std::milli>(histo_time).count();
            double gather_ms     = std::chrono::duration<double, std::milli>(gather_time).count();
            double reduce_ms     = std::chrono::duration<double, std::milli>(reduce_time).count();
            double entropy_ms    = std::chrono::duration<double, std::milli>(entropy_time).count();

            printf("Time to move to device:  %lf ms\n", transfer_ms);
            printf("Time to compute histograms: %lf ms\n", histo_ms);
            printf("Time to gather: %lf ms\n", gather_ms);
            printf("Time compute reduce: %lf ms\n", reduce_ms);
            printf("Time to compute H(X;Y):  %lf ms\n", entropy_ms);

            fprintf(csv_file, "%d,%lf,%lf,%lf,%lf,%lf\n",
                depth,
                transfer_ms,
                histo_ms,
                gather_ms,
                reduce_ms,
                entropy_ms
            );
        }
    }
    // Cleanup
    free(img1_host);
    free(img2_host);
    fclose(csv_file);

    return 0;
}
